#include "hip/hip_runtime.h"
//perceptron.cpp

#include <iostream>

using namespace std;

// Simple Perceptron class
class Perceptron 
{
private:
    double *w;	//weights
    double *x;	//inputs
    int n1;	//n1=n+1, x[0], .... x[n], x[0] is 1
public:
    Perceptron(int nx, double weights[]) 
    {
	n1 = nx;
	w =  new double[nx];
	for(int i = 0; i < n1; i++)
	  w[i] = weights[i];
    }

    //Activation function (Step function)
    int activation(double value) 
    {
        return value >= 0 ? 1 : 0;
    }

    // Predicted output 
    int predictedOutput(double xi[]) 
    {
	x = xi;
        double sum = 0;
        for (int i = 0; i < n1; ++i) 
            sum += x[i] * w[i];
        
	 int y1 = activation(sum);

         return y1;
    }

  // Train the perceptron
  // eta = learning rate
  // epochs = number of times of training 
  // numSamples = number of different input sets (x_data)
  // y is target output
  void train(double *x_data, double *y, double eta, int numSamples, int epochs)
  {
    double *x;

    for (int m = 0; m < epochs; m++){
      for (int k = 0; k < numSamples; k++) {
        x = x_data + k * n1;
        int y1 = predictedOutput( x );
        int error = y[k] - y1;

        // Update weights and bias
        for (int i = 0; i < n1; i++) {
          double dwi = eta * error * x[i];
          w[i] = w[i] + dwi;
        }
      }
    }
  }

    void printWeights()
    {
	cout << "\nPerceptron weights: ";
	for(int i = 0; i < n1; i++)
	  cout << "\n  w" << i <<": " << w[i];
    }

    ~Perceptron()
    {
	delete w;
    }
};


int main() 
{
    // Training data for AND gate, four sets of X
    double inputs[] = 
    {
        1, 0, 0,	//x0=1, x1=0, x2=1
        1, 0, 1,	//x0=1, x1=0, x2=1
        1, 1, 0,	//x0=1, x1=1, x2=0
        1, 1, 1		//x0=1, x1=1, x2=1
    };

    double y[] = {0, 0, 0, 1};	//target outputs (labels)
    double w[] = {1, 1, 1};

    string gates = " AND ";
    
    //Construct a perception with a bias and 2 inputs 
    Perceptron perceptron(3, w);
    
    // Train the perceptron
    perceptron.train(inputs, y, 0.1, 4, 10);

    // Test the perceptron
    double x[3];
    cout << "Testing Perceptron:" << endl;
    x[0] = 1;
    for (int i = 0; i < 4; i++) {
      x[1] = i & 1;
      x[2] = i >> 1;
      int output = perceptron.predictedOutput(x);
      cout << "  " <<  x[2] << gates << x[1] << " = " << output << endl;   
    }

    perceptron.printWeights();
    cout << endl << "Hello, AI World!" << endl;
    
    return 0;
}

	// Use one thread to update one weight in parallel.
	
//Activation function (Step function)
 __device__ __host__ int activate(double value)
{
        return value >= 0 ? 1 : 0;
}

// Predicted output 
__device__ __host__ int predict(double x[], double w[], int n1)
{
        double sum = 0; 
        for (int i = 0; i < n1; ++i)
            sum += x[i] * w[i];

         int y1 = activate(sum);

         return y1;
}

__global__ void trainPerceptron(double *x_data, double *y, double *w, double eta, int n,  
		   int numSamples, int epochs)
{
   double *x;
   int i = threadIdx.x;

   for (int m = 0; m < epochs; m++){ 
     for (int k = 0; k < numSamples; k++) {
        x = x_data + k * n; 
        int y1 = predict(w, x, n);
        int error = y[k] - y1;
        // Update weights and bias 
        double dwi = eta * error * x[i];
        w[i] = w[i] + dwi;
        __syncthreads();
     }
   }
}

// Simple Perceptron class
class Perceptron
{
private:
    double *w;  //weights
    double *x;  //inputs
    int n1;     //n1=n+1, x[0], .... x[n], x[0] is 1
    int wSize;  //size of all weights in bytes
    int numSamples; //number of different input sets (x_data)
    double *d_w;    //device memory to store weights
    double *d_y;    //device memory to sotre target outputs
    double *d_x_data;   //device memory to store samples data

public:
    Perceptron(int nx, int num, double weights[], double *y,  double inputs[])
    {
        n1 = nx;
        numSamples = num;
        wSize = n1 * sizeof(double);
        int samplesSize = numSamples * wSize;
        int outputsSize = numSamples * sizeof(double);
        w =  new double[n1];
        for(int i = 0; i < n1; i++)
          w[i] = weights[i];
        hipMalloc(&d_w, wSize);
        hipMalloc(&d_y, outputsSize);
       hipMalloc(&d_y, outputsSize);
        hipMalloc(&d_x_data, samplesSize);
        hipMemcpy(d_w, w, wSize, hipMemcpyHostToDevice);
        hipMemcpy(d_y, y, outputsSize, hipMemcpyHostToDevice);
        hipMemcpy(d_x_data, inputs, samplesSize, hipMemcpyHostToDevice);
    }

    //Activation function (Step function)
    int activation(double value)
    {
        return activate(value);
    }

    // Predicted output 
    int predictedOutput(double xi[])
    {
       int y1 = predict(xi, w, n1);

        return y1;

    }

    // Train the perceptron
    // eta = learning rate
    // epochs = number of times of training 
    // numSamples = number of different input sets (x_data)
    // y is target output
    void train(double eta, int epochs)
    {
       trainPerceptron<<<1, n1>>>(d_x_data, d_y, d_w, eta, n1, numSamples, epochs);
       hipDeviceSynchronize();
       hipMemcpy(w, d_w, wSize, hipMemcpyDeviceToHost);
    }

    void printWeights()
    {
        cout << "\nPerceptron weights: ";
        for(int i = 0; i < n1; i++)
          cout << "\n  w" << i <<": " << w[i];
    }

    ~Perceptron()
    {
        delete w;
        hipFree(d_w);
        hipFree(d_y);
        hipFree(d_x_data);
    }
};

int main()
{
  ....
   //Construct a perception with a bias and 2 inputs 
   // 4 data sets
   Perceptron perceptron(3, 4, w, y, inputs);

   // Train the perceptron with eta = 0.1, and 10 epochs
   perceptron.train(0.1, 10);
   ....
}
